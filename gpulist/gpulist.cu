
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <sstream>

int main(int argc, char* argv[]) {
    int devcount;
    hipGetDeviceCount(&devcount);

    printf("================Available Devices==============\n\n");
    for (int i = 0; i < devcount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("device:%d : %s, compute: %d_%d\n", i, prop.name, prop.major, prop.minor);
    }
    printf("===============================================\n\n");
}